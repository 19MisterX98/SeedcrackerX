#include "hip/hip_runtime.h"
#include <thread>
#include <vector>
#include <iostream>
#include <string>
#include <fstream>
#include <cstdlib>
#include <utility>
#include <mutex>
#include <map>
#include <algorithm>
#define _USE_MATH_DEFINES
#include <cmath>
#include <chrono>


uint64_t millis() {return (std::chrono::duration_cast< std::chrono::milliseconds >(std::chrono::system_clock::now().time_since_epoch())).count();}

#define GPU_ASSERT(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
        exit(code);
    }
}
#define SETGPU(gpuId) hipSetDevice(gpuId);\
	GPU_ASSERT(hipPeekAtLastError());\
	GPU_ASSERT(hipDeviceSynchronize());\
	GPU_ASSERT(hipPeekAtLastError());

#define SKIP 8
#define THREAD_SIZE 256LLU
#define BLOCK_SIZE (1LLU<<29)
#define BATCH_SIZE (THREAD_SIZE * BLOCK_SIZE * SKIP)


__managed__ uint32_t count = 0;
__managed__ uint64_t seedBuff[60000];

#define SEED_SPACE (1LLU<<48)

__global__ __launch_bounds__(THREAD_SIZE) void InitalFilter(const uint64_t offset) {
	uint64_t seed = (uint64_t)blockIdx.x * (uint64_t)blockDim.x * SKIP + (uint64_t)threadIdx.x  * SKIP + offset;

    if (((seed * 25214903917LLU)&((1LLU<<48)-1LLU)) < 225179967946752LLU) return;

THIS_STATEMENT_WILL_BE_REPLACED

}

int main() {
	SETGPU(0);
	std::ofstream outfile("output_seeds.dat", std::ofstream::binary);
	std::ofstream updates("updates.dat", std::ofstream::binary);
	for (uint64_t offset = 0; offset < SEED_SPACE; offset += BATCH_SIZE) {
		uint64_t start = millis();
		
		count = 0;
		InitalFilter<<<BLOCK_SIZE, THREAD_SIZE>>>(offset);
		GPU_ASSERT(hipPeekAtLastError());	
		GPU_ASSERT(hipDeviceSynchronize());
		GPU_ASSERT(hipPeekAtLastError());

		if (count != 0) {
            for (uint64_t index = 0; index < count; index++) {
                outfile << seedBuff[index] << std::endl;
                outfile.flush();
            }
		}
        uint64_t step1 = millis()-start;
        updates << "Finished gpu: time_millis:" << step1 << ", output: " << count << ", run:" << ((SEED_SPACE - offset)/BATCH_SIZE)  << std::endl;
        updates.flush();
		std::cout << "Finished gpu: time_millis:" << step1 << ", output: " << count << ", run:" << ((SEED_SPACE - offset)/BATCH_SIZE)  << std::endl;
	}
	outfile.close();
    updates.close();
	return 1;
}