#include "hip/hip_runtime.h"
#include <thread>
#include <vector>
#include <iostream>
#include <string>
#include <fstream>
#include <stdlib.h>
#include <utility>
#include <mutex>
#include <map>
#include <algorithm>
#define _USE_MATH_DEFINES
#include <math.h>
#include <chrono>


uint64_t millis() {return (std::chrono::duration_cast< std::chrono::milliseconds >(std::chrono::system_clock::now().time_since_epoch())).count();}

#define GPU_ASSERT(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
	fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
	exit(code);
  }
}

#define SETGPU(gpuId) hipSetDevice(gpuId);\
	GPU_ASSERT(hipPeekAtLastError());\
	GPU_ASSERT(hipDeviceSynchronize());\
	GPU_ASSERT(hipPeekAtLastError());

#define DEVICEABLE __host__ __device__



#define THREAD_SIZE 256LLU
#define BLOCK_SIZE (1LLU<<27) //(1LLU<<29)
#define BATCH_SIZE (THREAD_SIZE * BLOCK_SIZE)




__managed__ uint32_t count = 0;
__managed__ uint64_t seedBuff[60000000];


__managed__ uint32_t countOut = 0;
__managed__ uint64_t outputSeedBuff[6000];//Max seed output for the secondary filter

#define TREE_X INIT_TREE_INNER_X
#define TREE_Z INIT_TREE_INNER_Z



#define signed_seed_t int64_t

#define MODULUS (1LL << 48)
#define SQUARE_SIDE (MODULUS / 16)
#define X_TRANSLATE 0
#define Z_TRANSLATE 11
#define L00 7847617LL
#define L01 (-18218081LL)
#define L10 4824621LL
#define L11 24667315LL
#define LI00 (24667315.0 / 16)
#define LI01 (18218081.0 / 16)
#define LI10 (-4824621.0 / 16)
#define LI11 (7847617.0 / 16)

#define CONST_MIN(a, b) ((a) < (b) ? (a) : (b))
#define CONST_MIN4(a, b, c, d) CONST_MIN(CONST_MIN(a, b), CONST_MIN(c, d))
#define CONST_MAX(a, b) ((a) > (b) ? (a) : (b))
#define CONST_MAX4(a, b, c, d) CONST_MAX(CONST_MAX(a, b), CONST_MAX(c, d))
#define CONST_FLOOR(x) ((x) < (signed_seed_t) (x) ? (signed_seed_t) (x) - 1 : (signed_seed_t) (x))
#define CONST_CEIL(x) ((x) == (signed_seed_t) (x) ? (signed_seed_t) (x) : CONST_FLOOR((x) + 1))
#define CONST_LOWER(x, m, c) ((m) < 0 ? ((x) + 1 - (double) (c) / MODULUS) * (m) : ((x) - (double) (c) / MODULUS) * (m))
#define CONST_UPPER(x, m, c) ((m) < 0 ? ((x) - (double) (c) / MODULUS) * (m) : ((x) + 1 - (double) (c) / MODULUS) * (m))

// for a parallelogram ABCD https://media.discordapp.net/attachments/668607204009574411/671018577561649163/unknown.png
#define B_X LI00
#define B_Z LI10
#define C_X (LI00 + LI01)
#define C_Z (LI10 + LI11)
#define D_X LI01
#define D_Z LI11
#define LOWER_X CONST_MIN4(0, B_X, C_X, D_X)
#define LOWER_Z CONST_MIN4(0, B_Z, C_Z, D_Z)
#define UPPER_X CONST_MAX4(0, B_X, C_X, D_X)
#define UPPER_Z CONST_MAX4(0, B_Z, C_Z, D_Z)
#define ORIG_SIZE_X (UPPER_X - LOWER_X + 1)
#define SIZE_X CONST_CEIL(ORIG_SIZE_X - D_X)
#define SIZE_Z CONST_CEIL(UPPER_Z - LOWER_Z + 1)
#define TOTAL_WORK_SIZE (SIZE_X * SIZE_Z)

#define SEED_SPACE TOTAL_WORK_SIZE


__global__ __launch_bounds__(THREAD_SIZE) void InitalFilter(const uint64_t offset) {
	uint64_t idx = (((uint64_t)blockIdx.x * (uint64_t)blockDim.x + (uint64_t)threadIdx.x))+offset;

    signed_seed_t lattice_x = (int64_t)(idx%SIZE_X) + LOWER_X;
    signed_seed_t lattice_z = (int64_t)(idx/SIZE_X) + LOWER_Z;

    lattice_z += (B_X * lattice_z < B_Z * lattice_x) * SIZE_Z;
    if (D_X * lattice_z > D_Z * lattice_x) {
        lattice_x += B_X;
        lattice_z += B_Z;
    }

    lattice_x += (signed_seed_t) (TREE_X * LI00 + TREE_Z * LI01);
    lattice_z += (signed_seed_t) (TREE_X * LI10 + TREE_Z * LI11);

    uint64_t seed = (lattice_x * L00 + lattice_z * L01 + X_TRANSLATE)  & (MODULUS-1);

	PRIMARY_TREE_FILTER

	//TODO: Have different seed buffers per thread or somthing, so that the atomicAdd isnt a bottleneck
	seedBuff[atomicAdd(&count, 1)] = seed;
}





AUX_TREE_FUNCTIONS_REPLACEMENT



#define NEXT_INT_16(seed) (((seed = ((seed * 0x5DEECE66DLLU + 0xBLLU)&((1LLU<<48)-1)))>>(48-4)))

#define TREE_TEST(testMethod, index, expected_x, expected_z, IF_TYPE) IF_TYPE ((!(mask & (1<<index))) && x_pos == expected_x && z_pos == expected_z) mask |= ((uint8_t)testMethod(seed))<<index;
#define TARGET_MASK ((1<<AUXILIARY_TREE_COUNT)-1)
__global__ __launch_bounds__(THREAD_SIZE) void SecondaryFilter() {
	uint64_t idx = ((((uint64_t)blockIdx.x * (uint64_t)blockDim.x + (uint64_t)threadIdx.x)));
	if (idx >= count)
		return;
	uint64_t seed = seedBuff[idx];
	seed = LCG_REVERSE_STAGE_2_REPLACEMENT;
	
	uint8_t mask = 0;
	int32_t x_pos;
	int32_t z_pos = NEXT_INT_16(seed);
	for (int32_t index = 0; index < MAX_TREE_RNG_RANGE_REPLACEMENT * 2 && mask != TARGET_MASK; index++) {
        x_pos = z_pos;
		z_pos = NEXT_INT_16(seed);
		
        AUX_TREE_TEST_INNER_LOOP_CALL_REPLACEMENT
		

	}
	
	
	if (mask != TARGET_MASK)
		return;
	
	outputSeedBuff[atomicAdd(&countOut, 1)] = seedBuff[idx];
}











int main() {
	SETGPU(0);
	std::ofstream outfile("output_seeds.dat", std::ofstream::binary);
	for (uint64_t offset = 0; offset < SEED_SPACE; offset += BATCH_SIZE) {
		uint64_t start = millis();
		
		count = 0;
		countOut = 0;
		InitalFilter<<<BLOCK_SIZE, THREAD_SIZE>>>(offset);
		GPU_ASSERT(hipPeekAtLastError());	
		GPU_ASSERT(hipDeviceSynchronize());
		GPU_ASSERT(hipPeekAtLastError());
		uint64_t step1 = millis()-start;
		start = millis();

        uint64_t step2 = 0;
        uint64_t step3 = 0;
		if (count != 0) {
            SecondaryFilter<<<ceil((double)count/THREAD_SIZE), THREAD_SIZE>>>();
            GPU_ASSERT(hipPeekAtLastError());
            GPU_ASSERT(hipDeviceSynchronize());
            GPU_ASSERT(hipPeekAtLastError());
            step2 = millis()-start;
            start = millis();

            for (uint64_t index = 0; index < countOut; index++) {
                outfile << outputSeedBuff[index] << std::endl;
                outfile.flush();
            }
            step3 = millis()-start;
		}

		std::cout << "Finished gpu: " << (step1+step2+step3) << ", " << step1 << ", " << step2 << ", " << step3 << ", " << count << ", " << countOut << ", " << ((SEED_SPACE - offset)/BATCH_SIZE)  << std::endl;
	}
	outfile.close();
	return 1;
}